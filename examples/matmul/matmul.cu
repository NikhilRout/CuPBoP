
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

// Matrix multiplication kernel
__global__ void matrixMul(float *a, float *b, float *c, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row < N && col < N) {
        float sum = 0.0f;
        for (int i = 0; i < N; i++) {
            sum += a[row * N + i] * b[i * N + col];
        }
        c[row * N + col] = sum;
    }
}

int main() {
    int N = 32; // Matrix size N x N
    size_t size = N * N * sizeof(float);
    float *h_a, *h_b, *h_c; // Host matrices
    float *d_a, *d_b, *d_c; // Device matrices
    
    // Allocate host memory
    h_a = (float*)malloc(size);
    h_b = (float*)malloc(size);
    h_c = (float*)malloc(size);
    
    // Initialize matrices
    for (int i = 0; i < N * N; i++) {
        h_a[i] = 1.0f;
        h_b[i] = 2.0f;
    }
    
    // Allocate device memory
    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size);
    
    // Copy data to device
    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);
    
    // Set grid and block dimensions
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((N + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (N + threadsPerBlock.y - 1) / threadsPerBlock.y);
    
    // Launch kernel
    matrixMul<<<numBlocks, threadsPerBlock>>>(d_a, d_b, d_c, N);
    
    // Copy result back to host
    hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);
    
    // Verify result
    bool correct = true;
    for (int i = 0; i < N * N; i++) {
        if (h_c[i] != 2.0f * N) { // Expected value for this initialization
            correct = false;
            break;
        }
    }
    printf("%s\n", correct ? "PASS" : "FAIL");
    
    // Cleanup
    free(h_a);
    free(h_b);
    free(h_c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    
    return 0;
}